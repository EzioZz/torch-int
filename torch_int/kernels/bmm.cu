#include "include/bmm.h"
#include "include/common.h"
#include <cutlass/core_io.h>
#include <cutlass/cutlass.h>
#include <cutlass/gemm/device/gemm.h>
#include <cutlass/gemm/device/gemm_batched.h>
#include <cutlass/numeric_types.h>
#include <cutlass/util/host_tensor.h>
#include <hipblas.h>
#include <ATen/ATen.h>
#include <ATen/cuda/CUDABlas.h>
#include <ATen/cuda/Exceptions.h>

torch::Tensor bmm_s8t_s8n_f32t(torch::Tensor A, torch::Tensor B, float alpha) {
  int batch_size = A.size(0);
  int M = A.size(1);
  int N = B.size(1);
  int K = A.size(2);

  auto C = torch::empty({batch_size, M, N},
                        torch::dtype(torch::kFloat32).device(A.device()));
  int lda = A.size(2);
  int ldb = B.size(2);
  int ldc = C.size(2);

  using LayoutInputA = cutlass::layout::RowMajor;
  using LayoutInputB = cutlass::layout::ColumnMajor;
  using LayoutOutput = cutlass::layout::RowMajor;

  using ElementOutput = float;
  using ElementInputA = int8_t;
  using ElementInputB = int8_t;
  using ElementAccumulator = int32_t;
  using ElementComputeEpilogue = float;

  using EpilogueOp = cutlass::epilogue::thread::LinearCombination<
      ElementOutput, 128 / cutlass::sizeof_bits<ElementOutput>::value,
      ElementAccumulator, ElementComputeEpilogue>;

  using Gemm = cutlass::gemm::device::GemmBatched<
      ElementInputA, LayoutInputA, ElementInputB, LayoutInputB, ElementOutput,
      LayoutOutput, ElementAccumulator, cutlass::arch::OpClassTensorOp,
      cutlass::arch::Sm80, cutlass::gemm::GemmShape<256, 128, 64>,
      cutlass::gemm::GemmShape<64, 64, 64>, cutlass::gemm::GemmShape<16, 8, 32>,
      EpilogueOp>;

  long long int batch_stride_A = M * K;
  long long int batch_stride_B = N * K;
  long long int batch_stride_C = M * N;

  Gemm gemm_op;
  typename Gemm::Arguments arguments{
      {M, N, K},      {A.data_ptr<ElementInputA>(), lda},
      batch_stride_A, {B.data_ptr<ElementInputB>(), ldb},
      batch_stride_B, {C.data_ptr<ElementOutput>(), ldc},
      batch_stride_C, {C.data_ptr<ElementOutput>(), ldc},
      batch_stride_C, {alpha, 0},
      batch_size};

  // Using the arguments, query for extra workspace required for matrix
  // multiplication computation
  size_t workspace_size = Gemm::get_workspace_size(arguments);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  // Check the problem size is supported or not
  cutlass::Status status = gemm_op.can_implement(arguments);
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error("cutlass cannot implement");
  }

  // Initialize CUTLASS kernel with arguments and workspace pointer
  status = gemm_op.initialize(arguments, workspace.get());
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error("cutlass cannot initialize");
  }

  status = gemm_op();
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error("cutlass cannot run");
  }
  return C;
}

torch::Tensor bmm_s8t_s8n_s8t(torch::Tensor A, torch::Tensor B, float alpha) {
  int batch_size = A.size(0);
  int M = A.size(1);
  int N = B.size(1);
  int K = A.size(2);

  auto C = torch::empty({batch_size, M, N},
                        torch::dtype(torch::kInt8).device(A.device()));
  int lda = A.size(2);
  int ldb = B.size(2);
  int ldc = C.size(2);

  using LayoutInputA = cutlass::layout::RowMajor;
  using LayoutInputB = cutlass::layout::ColumnMajor;
  using LayoutOutput = cutlass::layout::RowMajor;

  using ElementOutput = int8_t;
  using ElementInputA = int8_t;
  using ElementInputB = int8_t;
  using ElementAccumulator = int32_t;
  using ElementComputeEpilogue = float;

  using EpilogueOp = cutlass::epilogue::thread::LinearCombinationClamp<
      ElementOutput, 128 / cutlass::sizeof_bits<ElementOutput>::value,
      ElementAccumulator, ElementComputeEpilogue>;

  using Gemm = cutlass::gemm::device::GemmBatched<
      ElementInputA, LayoutInputA, ElementInputB, LayoutInputB, ElementOutput,
      LayoutOutput, ElementAccumulator, cutlass::arch::OpClassTensorOp,
      cutlass::arch::Sm80, cutlass::gemm::GemmShape<256, 128, 64>,
      cutlass::gemm::GemmShape<64, 64, 64>, cutlass::gemm::GemmShape<16, 8, 32>,
      EpilogueOp>;

  long long int batch_stride_A = M * K;
  long long int batch_stride_B = N * K;
  long long int batch_stride_C = M * N;

  Gemm gemm_op;
  typename Gemm::Arguments arguments{
      {M, N, K},      {A.data_ptr<ElementInputA>(), lda},
      batch_stride_A, {B.data_ptr<ElementInputB>(), ldb},
      batch_stride_B, {C.data_ptr<ElementOutput>(), ldc},
      batch_stride_C, {C.data_ptr<ElementOutput>(), ldc},
      batch_stride_C, {alpha, 0},
      batch_size};

  // Using the arguments, query for extra workspace required for matrix
  // multiplication computation
  size_t workspace_size = Gemm::get_workspace_size(arguments);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  // Check the problem size is supported or not
  cutlass::Status status = gemm_op.can_implement(arguments);
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error("cutlass cannot implement");
  }

  // Initialize CUTLASS kernel with arguments and workspace pointer
  status = gemm_op.initialize(arguments, workspace.get());
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error("cutlass cannot initialize");
  }

  status = gemm_op();
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error("cutlass cannot run");
  }
  return C;
}

torch::Tensor bmm_s8t_s8n_s32t(torch::Tensor A, torch::Tensor B) {
  int batch_size = A.size(0);
  int M = A.size(1);
  int N = B.size(1);
  int K = A.size(2);

  auto C = torch::empty({batch_size, M, N},
                        torch::dtype(torch::kInt32).device(A.device()));
  int lda = A.size(2);
  int ldb = B.size(2);
  int ldc = C.size(2);

  using LayoutInputA = cutlass::layout::RowMajor;
  using LayoutInputB = cutlass::layout::ColumnMajor;
  using LayoutOutput = cutlass::layout::RowMajor;

  using ElementOutput = int32_t;
  using ElementInputA = int8_t;
  using ElementInputB = int8_t;
  using ElementAccumulator = int32_t;
  using ElementComputeEpilogue = int32_t;

  using EpilogueOp = cutlass::epilogue::thread::LinearCombination<
      ElementOutput, 128 / cutlass::sizeof_bits<ElementOutput>::value,
      ElementAccumulator, ElementComputeEpilogue>;

  using Gemm = cutlass::gemm::device::GemmBatched<
      ElementInputA, LayoutInputA, ElementInputB, LayoutInputB, ElementOutput,
      LayoutOutput, ElementAccumulator, cutlass::arch::OpClassTensorOp,
      cutlass::arch::Sm80, cutlass::gemm::GemmShape<256, 128, 64>,
      cutlass::gemm::GemmShape<64, 64, 64>, cutlass::gemm::GemmShape<16, 8, 32>,
      EpilogueOp>;

  long long int batch_stride_A = M * K;
  long long int batch_stride_B = N * K;
  long long int batch_stride_C = M * N;

  Gemm gemm_op;

  ElementComputeEpilogue alpha = 1;

  cutlass::Status status = gemm_op({{M, N, K},
                                    {A.data_ptr<ElementInputA>(), lda},
                                    batch_stride_A,
                                    {B.data_ptr<ElementInputB>(), ldb},
                                    batch_stride_B,
                                    {C.data_ptr<ElementOutput>(), ldc},
                                    batch_stride_C,
                                    {C.data_ptr<ElementOutput>(), ldc},
                                    batch_stride_C,
                                    {alpha, 0},
                                    batch_size});

  if (status != cutlass::Status::kSuccess) {
    std::cout << "cutlass error code: " << (int)status << std::endl;
  }
  return C;
}

torch::Tensor bmm_s8t_s8n_s32t_cublas(torch::Tensor A, torch::Tensor B) {
  int batch_size = A.size(0);
  int M = A.size(1);
  int N = B.size(1); // b, N, K
  int K = A.size(2); 
  auto C = torch::empty({batch_size, M, N},
                        torch::dtype(torch::kInt32).device(A.device()));

  int lda = A.size(2); // K
  int ldb = B.size(2); // N
  int ldc = C.size(2); // N

  // using LayoutInputA = cutlass::layout::RowMajor;
  // using LayoutInputB = cutlass::layout::ColumnMajor;
  // using LayoutOutput = cutlass::layout::RowMajor;

  // using ElementOutput = int32_t;
  // using ElementInputA = int8_t;
  // using ElementInputB = int8_t;

  long long int batch_stride_A = M * K;
  long long int batch_stride_B = N * K;
  long long int batch_stride_C = M * N;

  hipblasStatus_t status;

  hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();


  int32_t alpha = 1; 
  int32_t beta = 0; 

  status = hipblasGemmStridedBatchedEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                                    N, M, K,
                                      (const void*)&alpha, 
                                      (const void*)B.data_ptr(), HIP_R_8I, K, batch_stride_B,
                                      (const void*)A.data_ptr(), HIP_R_8I, K, batch_stride_A,
                                      (const void*)&beta,
                                      (void*)C.data_ptr(), HIP_R_32I, N, batch_stride_C,
                                      batch_size,
                                      HIP_R_32I, 
                                      CUBLAS_GEMM_DEFAULT_TENSOR_OP);
  if (status != HIPBLAS_STATUS_SUCCESS) {
      printf("cuBLAS API failed with status %d\n", status);
  }


  return C;
  // hipblasStatus_t hipblasGemmStridedBatchedEx(hipblasHandle_t handle,
  //                             hipblasOperation_t transa,
  //                             hipblasOperation_t transb,
  //                             int m,
  //                             int n,
  //                             int k,
  //                             const void    *alpha,
  //                             const void     *A,
  //                             hipDataType Atype,
  //                             int lda,
  //                             long long int strideA,
  //                             const void     *B,
  //                             hipDataType Btype,
  //                             int ldb,
  //                             long long int strideB,
  //                             const void    *beta,
  //                             void           *C,
  //                             hipDataType Ctype,
  //                             int ldc,
  //                             long long int strideC,
  //                             int batchCount,
  //                             hipDataType computeType,
  //                             hipblasGemmAlgo_t algo)

}


torch::Tensor bmm_s8t_s8n_s32t_cublas1batch(torch::Tensor A, torch::Tensor B) {
  int batch_size = A.size(0);
  int M = A.size(1);
  int N = B.size(1); // b, N, K
  int K = A.size(2); 
  auto C = torch::empty({batch_size, M, N},
                        torch::dtype(torch::kInt32).device(A.device()));

  int lda = A.size(2); // K
  int ldb = B.size(2); // N
  int ldc = C.size(2); // N

  // using LayoutInputA = cutlass::layout::RowMajor;
  // using LayoutInputB = cutlass::layout::ColumnMajor;
  // using LayoutOutput = cutlass::layout::RowMajor;

  // using ElementOutput = int32_t;
  // using ElementInputA = int8_t;
  // using ElementInputB = int8_t;

  long long int batch_stride_A = M * K;
  long long int batch_stride_B = N * K;
  long long int batch_stride_C = M * N;

  hipblasStatus_t status;

  hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();


  int32_t alpha = 1; 
  int32_t beta = 0; 

  status = hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                                    N, M, K,
                                      (const void*)&alpha, 
                                      (const void*)B.data_ptr(), HIP_R_8I, K,
                                      (const void*)A.data_ptr(), HIP_R_8I, K,
                                      (const void*)&beta,
                                      (void*)C.data_ptr(), HIP_R_32I, N,
                                      HIP_R_32I, 
                                      CUBLAS_GEMM_DEFAULT_TENSOR_OP);
  if (status != HIPBLAS_STATUS_SUCCESS) {
      printf("cuBLAS API failed with status %d\n", status);
  }


  return C;
  // hipblasStatus_t hipblasGemmStridedBatchedEx(hipblasHandle_t handle,
  //                             hipblasOperation_t transa,
  //                             hipblasOperation_t transb,
  //                             int m,
  //                             int n,
  //                             int k,
  //                             const void    *alpha,
  //                             const void     *A,
  //                             hipDataType Atype,
  //                             int lda,
  //                             long long int strideA,
  //                             const void     *B,
  //                             hipDataType Btype,
  //                             int ldb,
  //                             long long int strideB,
  //                             const void    *beta,
  //                             void           *C,
  //                             hipDataType Ctype,
  //                             int ldc,
  //                             long long int strideC,
  //                             int batchCount,
  //                             hipDataType computeType,
  //                             hipblasGemmAlgo_t algo)

}